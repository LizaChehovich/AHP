#include "hip/hip_runtime.h"
#include "helper.h"
#include "image_helper.h"
#include "constant.h"
//#include "cpu_functions.h"
//#include "host_cuda_functions.h"

#include <cmath>


using namespace std;

double gpu_convert_image(uint8_t* input, uint8_t* result, unsigned int width, unsigned int height, unsigned int channels);
double gpu_stream_convert_image(uint8_t* input, uint8_t* result, unsigned int width, unsigned int height, unsigned int channels);

double cpu_convert_image(uint8_t* input, uint8_t* result, unsigned int width, unsigned int height, unsigned int chnnels);

double cpu_and_gpu_convert_image(uint8_t* input, uint8_t* result, unsigned int width, unsigned int height, unsigned int chnnels);

__device__ void gray_center_kernel(const int* input, int* result, const int width, const int heigth, const int pitch, const int res_pitch)
{

	const int count_row = Xdim + 2;
	const int count_column = Ydim + 2;
	const int pitch_in_int = pitch / sizeof(int);
	const int absX = blockIdx.x * blockDim.x + threadIdx.x;
	const int absY = blockIdx.y * blockDim.y + threadIdx.y;

	const int absXinByte = absX * sizeof(int);

	const int Width = width + 2 * sizeof(int);
	const int Height = heigth + 2;

	__shared__ int buf[count_column][count_row];

	//load data in shared memory

	buf[threadIdx.y][threadIdx.x] = input[absY*pitch_in_int + absX];

	if (threadIdx.y < 2)
		buf[threadIdx.y + blockDim.y][threadIdx.x] = input[(absY + blockDim.y)*pitch_in_int + absX];

	if (threadIdx.x < 2)
		buf[threadIdx.y][count_row - 2 + threadIdx.x] = input[absY*pitch_in_int + absX + blockDim.x];

	if (threadIdx.y < 2  && threadIdx.x < 2)
		buf[threadIdx.y + blockDim.y][count_row - 2 + threadIdx.x] = input[(absY + blockDim.y)*pitch_in_int + absX + blockDim.x];

	__syncthreads();

	int val = 0;
	float sum;
	int bytePosY = threadIdx.y + 1;
	int bytePosX = (threadIdx.x + 1) * 4;

	for (int k = 0; k < 4; k++)
	{
		sum = 0;

		for (int i = -1; i < 2; i++)
		{
			for (int j = -1; j < 2; j++)
			{
				sum += ((byte*)buf[bytePosY + i])[bytePosX + j] * d_filter[i + 1][j + 1];
			}
		}
		sum = round(sum);
		if (sum < 0) sum = 0;
		if (sum > 255) sum = 255;

		((byte*)&val)[k] = sum;
		bytePosX++;
	}

	result[absY*res_pitch / sizeof(int) + absX] = val;
}

__device__ void gray_frame_kernel(const int* input, int* result, const int width, const int heigth, const int pitch, const int res_pitch)
{
	const int count_row = Xdim + 2;
	const int count_column = Ydim + 2;
	const int pitch_in_int = pitch / sizeof(int);
	const int absX = blockIdx.x * blockDim.x + threadIdx.x;
	const int absY = blockIdx.y * blockDim.y + threadIdx.y;

	const int absXinByte = absX * sizeof(int);

	const int Width = width + 2 * sizeof(int);
	const int Height = heigth + 2;

	__shared__ int buf[count_column][count_row];

	//load data in shared memory

	if (absY < Height && absXinByte < Width)
		buf[threadIdx.y][threadIdx.x] = input[absY*pitch_in_int + absX];

	if (threadIdx.y < 2 && ((absY + blockDim.y) < Height) && absXinByte < Width)
		buf[threadIdx.y + blockDim.y][threadIdx.x] = input[(absY + blockDim.y)*pitch_in_int + absX];

	if (absY < Height && threadIdx.x < 2 && ((absX + blockDim.x) * sizeof(int)) < Width)
		buf[threadIdx.y][count_row - 2 + threadIdx.x] = input[absY*pitch_in_int + absX + blockDim.x];

	if (threadIdx.y < 2 && (absY + blockDim.y) < Height && threadIdx.x < 2 && ((absX + blockDim.x) * sizeof(int)) < Width)
		buf[threadIdx.y + blockDim.y][count_row - 2 + threadIdx.x] = input[(absY + blockDim.y)*pitch_in_int + absX + blockDim.x];

	__syncthreads();

	int val = 0;
	float sum;
	int bytePosY = threadIdx.y + 1;
	int bytePosX = (threadIdx.x + 1) * 4;

	for (int k = 0; k < 4; k++)
	{
		if (absY >= Height && absXinByte >= Width)
			break;
		sum = 0;

		for (int i = -1; i < 2; i++)
		{
			for (int j = -1; j < 2; j++)
			{
				sum += ((byte*)buf[bytePosY + i])[bytePosX + j] * d_filter[i + 1][j + 1];
			}
		}
		sum = round(sum);
		if (sum < 0) sum = 0;
		if (sum > 255) sum = 255;

		((byte*)&val)[k] = sum;
		bytePosX++;
	}

	if (absY < heigth && absXinByte < width)
	{
		result[absY*res_pitch / sizeof(int) + absX] = val;
	}
}

__global__ void cuda_gray_processing(const int* input, int* result, const int width, const int heigth, const int pitch, const int res_pitch)
{
	if (blockIdx.x == gridDim.x - 1 || blockIdx.y == gridDim.y - 1)
		gray_frame_kernel(input, result, width, heigth, pitch, res_pitch);
	else
		gray_center_kernel(input, result, width, heigth, pitch, res_pitch);
}

__global__ void cuda_gray_processing(const int* input, int* result, const int width, const int heigth, const int pitch, const int res_pitch, const int frame)
{
	if (frame)
		if (blockIdx.x == gridDim.x - 1 || blockIdx.y == gridDim.y - 1)
			gray_frame_kernel(input, result, width, heigth, pitch, res_pitch);
		else
			gray_center_kernel(input, result, width, heigth, pitch, res_pitch);
	else
		gray_center_kernel(input, result, width, heigth, pitch, res_pitch);
}

__device__ void color_center_kernel(const int* input, int* result, const int width, const int heigth, const int pitch, const int res_pitch)
{
	const int channels = 3;
	const int count_row = (Xdim*channels + 2);
	const int count_column = Ydim + 2;
	const int pitch_in_int = pitch / sizeof(int);
	const int absX = (blockIdx.x * blockDim.x)*channels + threadIdx.x;
	const int absY = blockIdx.y * blockDim.y + threadIdx.y;

	const int absXinByte = absX * sizeof(int);
	const int blockDimXinByte = blockDim.x * sizeof(int);

	const int Width = (width * channels + 2 * sizeof(int));
	const int Height = heigth + 2;

	__shared__ int buf[count_column][count_row];

	//load data in shared memory

	for (int i = 0; i < channels; i++) {
		buf[threadIdx.y][threadIdx.x + blockDim.x*i] = input[absY*pitch_in_int + absX + blockDim.x*i];

		buf[threadIdx.y + blockDim.y][threadIdx.x + blockDim.x*i] = input[(absY + blockDim.y)*pitch_in_int + absX + blockDim.x*i];
	}

	if (threadIdx.x < 2)
		buf[threadIdx.y][count_row - 2 + threadIdx.x] = input[absY*pitch_in_int + absX + blockDim.x*channels];

	if (threadIdx.y < 2 && threadIdx.x < 2)
		buf[threadIdx.y + blockDim.y][count_row - 2 + threadIdx.x] = input[(absY + blockDim.y)*pitch_in_int + absX + blockDim.x*channels];

	__syncthreads();

	int val;
	float sum;

	for (int c = 0; c < channels; c++)
	{

		val = 0;

		int bytePosY = threadIdx.y + 1;
		int bytePosX = (threadIdx.x + 1 + c*blockDim.x) * sizeof(int);

		for (int k = 0; k < 4; k++)
		{
			sum = 0;

			for (int i = -1; i < 2; i++)
			{
				for (int j = -1; j < 2; j++)
				{
					sum += ((byte*)buf[bytePosY + i])[bytePosX + j*channels] * d_filter[i + 1][j + 1];
				}
			}
			sum = round(sum);
			if (sum < 0) sum = 0;
			if (sum > 255) sum = 255;

			((byte*)&val)[k] = sum;
			bytePosX++;
		}

		result[absY*res_pitch / sizeof(int) + absX + c*blockDim.x] = val;
	}
}

__device__ void color_frame_kernel(const int* input, int* result, const int width, const int heigth, const int pitch, const int res_pitch)
{
	const int channels = 3;
	const int count_row = (Xdim*channels + 2);
	const int count_column = Ydim + 2;
	const int pitch_in_int = pitch / sizeof(int);
	const int absX = (blockIdx.x * blockDim.x)*channels + threadIdx.x;
	const int absY = blockIdx.y * blockDim.y + threadIdx.y;

	const int absXinByte = absX * sizeof(int);
	const int blockDimXinByte = blockDim.x * sizeof(int);

	const int Width = (width * channels + 2 * sizeof(int));
	const int Height = heigth + 2;

	__shared__ int buf[count_column][count_row];

	//load data in shared memory

	for (int i = 0; i < channels; i++) {
		if (absY < Height && absXinByte + blockDimXinByte*i < Width)
			buf[threadIdx.y][threadIdx.x + blockDim.x*i] = input[absY*pitch_in_int + absX + blockDim.x*i];

		if (threadIdx.y < 2 && ((absY + blockDim.y) < Height) && absXinByte + blockDimXinByte*i < Width)
			buf[threadIdx.y + blockDim.y][threadIdx.x + blockDim.x*i] = input[(absY + blockDim.y)*pitch_in_int + absX + blockDim.x*i];
	}

	if (absY < Height && threadIdx.x < 2 && ((absX + blockDim.x*channels) * sizeof(int)) < Width)
		buf[threadIdx.y][count_row - 2 + threadIdx.x] = input[absY*pitch_in_int + absX + blockDim.x*channels];

	if (threadIdx.y < 2 && (absY + blockDim.y) < Height && threadIdx.x < 2 && ((absX + blockDim.x*channels) * sizeof(int)) < Width)
		buf[threadIdx.y + blockDim.y][count_row - 2 + threadIdx.x] = input[(absY + blockDim.y)*pitch_in_int + absX + blockDim.x*channels];

	__syncthreads();

	int val;
	float sum;

	for (int c = 0; c < channels; c++)
	{

		val = 0;

		int bytePosY = threadIdx.y + 1;
		int bytePosX = (threadIdx.x + 1 + c*blockDim.x) * sizeof(int);

		for (int k = 0; k < 4; k++)
		{
			if (absY >= Height && absXinByte + c*blockDim.x * sizeof(int) >= Width)
				break;
			sum = 0;

			for (int i = -1; i < 2; i++)
			{
				for (int j = -1; j < 2; j++)
				{
					sum += ((byte*)buf[bytePosY + i])[bytePosX + j*channels] * d_filter[i + 1][j + 1];
				}
			}
			sum = round(sum);
			if (sum < 0) sum = 0;
			if (sum > 255) sum = 255;

			((byte*)&val)[k] = sum;
			bytePosX++;
		}

		if (absY < heigth && absXinByte + c * blockDimXinByte < width * channels)
		{
			result[absY*res_pitch / sizeof(int) + absX + c*blockDim.x] = val;
		}
	}
}

__global__ void cuda_color_processing(const int* input, int* result, const int width, const int heigth, const int pitch, const int res_pitch)
{
	if (blockIdx.x == gridDim.x - 1 || blockIdx.y == gridDim.y - 1)
		color_frame_kernel(input, result, width, heigth, pitch, res_pitch);
	else
		color_center_kernel(input, result, width, heigth, pitch, res_pitch);
}

__global__ void cuda_color_processing(const int* input, int* result, const int width, const int heigth, const int pitch, const int res_pitch, const int frame)
{
	if (frame)
		if (blockIdx.x == gridDim.x - 1 || blockIdx.y == gridDim.y - 1)
			color_frame_kernel(input, result, width, heigth, pitch, res_pitch);
		else
			color_center_kernel(input, result, width, heigth, pitch, res_pitch);
	else
		color_center_kernel(input, result, width, heigth, pitch, res_pitch);
}

int main()
{
	return menu();
}

double gpu_convert_image(uint8_t* input, uint8_t* result, unsigned int width, unsigned int height, unsigned int channels)
{
	hipMemcpyToSymbol(HIP_SYMBOL(d_filter), filter, 9 * sizeof(float), 0, hipMemcpyHostToDevice);

	uint8_t* dev_input;
	uint8_t* dev_output;
	float time = 0;

	size_t input_pitch;
	size_t res_pitch;

	hipError_t err = hipSuccess;

	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);


	dim3 threadsPerBlock(Xdim, Ydim);
	dim3 numBlocks(ceil(((float)width + Xdim - 1) / Xdim / 4), ceil((height + Ydim - 1) / Ydim));

	hipEventRecord(start);

	err = hipMallocPitch(&dev_input, &input_pitch, (width * channels + 2 * sizeof(int)), height + 2);
	if (error(err))		return 0.0;
	err = hipMemset2D(dev_input, input_pitch, 0, (width * channels + 2 * sizeof(int)), height + 2);
	if (error(err))		return 0.0;
	err = hipMemcpy2D(dev_input + input_pitch + sizeof(int), input_pitch, input, width * channels, width * channels * sizeof(uint8_t), height, hipMemcpyHostToDevice);
	if (error(err))		return 0.0;
	err = hipMallocPitch(&dev_output, &res_pitch, width * channels, height);
	if (error(err))		return 0.0;

	if (channels == 3)
		cuda_color_processing << <numBlocks, threadsPerBlock >> > (reinterpret_cast<int*>(dev_input), reinterpret_cast<int*>(dev_output), width, height, input_pitch, res_pitch);
	else
		cuda_gray_processing << <numBlocks, threadsPerBlock >> > (reinterpret_cast<int*>(dev_input), reinterpret_cast<int*>(dev_output), width, height, input_pitch, res_pitch);


	err = hipMemcpy2D(result, width * channels, dev_output, res_pitch, width * channels * sizeof(uint8_t), height, hipMemcpyDeviceToHost);
	if (error(err))		return 0.0;

	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&time, start, end);

	hipFree(dev_input);
	hipFree(dev_output);
	hipEventDestroy(start);
	hipEventDestroy(end);
	return time;
}

double gpu_stream_convert_image(uint8_t* input, uint8_t* result, unsigned int width, unsigned int height, unsigned int channels)
{
	hipMemcpyToSymbol(HIP_SYMBOL(d_filter), filter, 9 * sizeof(float), 0, hipMemcpyHostToDevice);

	float time = 0;
	int stream_height = ceil(((float)height) / Ydim / CountStream)*Ydim;
	int Width = width*channels;
	
	size_t input_pitch;
	size_t res_pitch;

	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);

	hipError_t err = hipSuccess;

	hipStream_t stream[CountStream];

	uint8_t* dev_input[CountStream];
	uint8_t* dev_output[CountStream];

	err = hipHostRegister(input, Width*height, hipHostRegisterPortable);
	if (error(err))		return 0.0;

	err = hipHostRegister(result, Width*height, hipHostRegisterPortable);
	if (error(err))		return 0.0;

	int height_count;

	for(int i = 0; i< CountStream; i++)
		err = hipStreamCreate(&stream[i]);

	hipEventRecord(start);

	for (int i = 0; i < CountStream; i++)
	{
		height_count = (height > stream_height*(i + 1)) ? stream_height : height - stream_height*i;
		err = hipMallocPitch(&dev_input[i], &input_pitch, (Width + 2 * sizeof(int)), height_count + 2);
		err = hipMemset2DAsync(dev_input[i], input_pitch, 0, (Width + 2 * sizeof(int)), height_count + 2, stream[i]);
		err = hipMallocPitch(&dev_output[i], &res_pitch, Width, height_count);
		err = hipMemcpy2DAsync(dev_input[i] + input_pitch + sizeof(int), input_pitch, input + stream_height*i*Width, Width, Width * sizeof(uint8_t), height_count, hipMemcpyHostToDevice, stream[i]);
		if (i > 0)
		{
			err = hipMemcpy2DAsync(dev_input[i] + sizeof(int), input_pitch, input + stream_height*i * Width - Width, Width, Width * sizeof(uint8_t), 1, hipMemcpyHostToDevice, stream[i]);
		}
		if (i < CountStream - 1)
		{
			err = hipMemcpy2DAsync(dev_input[i] + input_pitch + sizeof(int) + height_count*input_pitch, input_pitch, input + stream_height*(i + 1) *Width, Width, Width * sizeof(uint8_t), 1, hipMemcpyHostToDevice, stream[i]);
		}
	}

	int frame;

	for (int i = 0; i < CountStream; i++)
	{
		height_count = (height > stream_height*(i + 1)) ? stream_height : height - stream_height*i;

		dim3 threadsPerBlock(Xdim, Ydim);
		dim3 numBlocks(ceil(((float)width + Xdim - 1) / Xdim / 4), ceil((float)height_count / Ydim));

		frame = i == CountStream - 1 ? 1 : 0;

		if (channels == 3)
			cuda_color_processing << <numBlocks, threadsPerBlock,0,stream[i] >> > (reinterpret_cast<int*>(dev_input[i]), reinterpret_cast<int*>(dev_output[i]), width, height_count, input_pitch, res_pitch);
		else
			cuda_gray_processing << <numBlocks, threadsPerBlock,0,stream[i] >> > (reinterpret_cast<int*>(dev_input[i]), reinterpret_cast<int*>(dev_output[i]), width, height_count, input_pitch, res_pitch, frame);
	}

	for (int i = 0; i < CountStream; i++)
	{
		height_count = (height > stream_height*(i + 1)) ? stream_height : height - stream_height*i;

		//err = hipStreamSynchronize(stream[i]);

		err = hipMemcpy2DAsync(result + stream_height*i*Width, Width, dev_output[i], res_pitch, Width * sizeof(uint8_t), height_count, hipMemcpyDeviceToHost, stream[i]);
	}

	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&time, start, end);

	for (int i = 0; i < CountStream; i++)
	{
		hipFree(dev_input[i]);
		hipFree(dev_output[i]);
	}
	
	err = hipHostUnregister(input);
	if (error(err))		return 0.0;

	err = hipHostUnregister(result);
	if (error(err))		return 0.0;

	hipEventDestroy(start);
	hipEventDestroy(end);
	return time;
}

double cpu_convert_image(uint8_t * input, uint8_t * result, unsigned int width, unsigned int height, unsigned int channels)
{
	float val;
	LARGE_INTEGER start, finish, freq;
	QueryPerformanceFrequency(&freq);
	QueryPerformanceCounter(&start);
	for (int x = 0; x < width * channels; x++)
	{
		for (int y = 0; y < height; y++)
		{
			val = 0;
			for (int i = -1; i < 2; i++)
			{
				for (int j = -1; j < 2; j++)
				{
					if (x + j * channels < 0 || x + j * channels >= width*channels || y + i < 0 || y + i >= height)
						continue;
					val += input[(y + i)*width * channels + (x + j * channels)] * filter[i + 1][j + 1];
				}
			}
			val = round(val);
			result[y * width * channels + x] = val<MIN ? MIN : val>MAX ? MAX : val;
		}
	}
	QueryPerformanceCounter(&finish);
	return (finish.QuadPart - start.QuadPart) * 1000 / (double)freq.QuadPart;
}

double cpu_and_gpu_convert_image(uint8_t * input, uint8_t * result, unsigned int width, unsigned int height, unsigned int channels)
{
	if (channels == 1)
		return gpu_convert_image(input, result, width, height, channels);

	hipMemcpyToSymbol(HIP_SYMBOL(d_filter), filter, 9 * sizeof(float), 0, hipMemcpyHostToDevice);

	float cuda_time = 0.0;
	int data_size = width*height;

	hipError_t err = hipSuccess;
	size_t input_pitch;
	size_t res_pitch;

	uint8_t* color[3];
	uint8_t* r_color[3];

	uint8_t* d_color[3];
	uint8_t* d_r_color[3];

	hipStream_t stream[3];

	for (int i = 0; i < channels; i++) {
		hipStreamCreate(&stream[i]);
		hipHostMalloc((void**)&color[i], data_size);
		hipHostMalloc((void**)&r_color[i], data_size);
		hipMallocPitch(&d_color[i], &input_pitch, width + 2 * sizeof(int), height + 2);
		hipMemset2DAsync(d_color[i], input_pitch, 0, width + 2 * sizeof(int), height + 2);
		hipMallocPitch(&d_r_color[i], &res_pitch, width, height);
	}

	dim3 threadsPerBlock(Xdim, Ydim);
	dim3 numBlocks(ceil(((float)width + Xdim - 1) / Xdim / 4), ceil((height + Ydim - 1) / Ydim));
	
	hipEvent_t begin, end;
	hipEventCreate(&begin);
	hipEventCreate(&end);

	LARGE_INTEGER start, finish, freq;
	QueryPerformanceFrequency(&freq);
	QueryPerformanceCounter(&start);

	for (int x = 0; x < width; x++)
		for (int y = 0; y < height; y++)
		{
			color[0][y*width + x] = input[(y*width + x)*channels];
			color[1][y*width + x] = input[(y*width + x)*channels + 1];
			color[2][y*width + x] = input[(y*width + x)*channels + 2];
		}

	hipEventRecord(begin);

	for (int i = 0; i < channels; i++)
	{
		hipMemcpy2DAsync(d_color[i] + input_pitch + sizeof(int), input_pitch, color[i], width, width * sizeof(uint8_t), height, hipMemcpyHostToDevice, stream[i]);
	}

	for (int i = 0; i < channels; i++)
	{
		cuda_gray_processing << <numBlocks, threadsPerBlock, 0, stream[i] >> >(reinterpret_cast<int*>(d_color[i]), reinterpret_cast<int*>(d_r_color[i]), width, height, input_pitch, res_pitch);
	}

	for(int i = 0; i <channels; i++)
	{
		hipMemcpy2DAsync(r_color[i], width, d_r_color[i], res_pitch, width * sizeof(uint8_t), height, hipMemcpyDeviceToHost, stream[i]);
	}

	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&cuda_time, begin, end);

	for (int x = 0; x < width; x++)
		for (int y = 0; y < height; y++)
		{
			result[(y*width + x)*channels] = r_color[0][y*width + x];
			result[(y*width + x)*channels + 1] = r_color[1][y*width + x];
			result[(y*width + x)*channels + 2] = r_color[2][y*width + x];
		}

	QueryPerformanceCounter(&finish);

	cout << "Cuda kernel time " << cuda_time << endl;

	for (int i = 0; i < channels; i++)
	{
		hipFree(d_color[i]);
		hipFree(d_r_color[i]);
		hipHostFree(color[i]);
		hipHostFree(r_color[i]);
		hipStreamDestroy(stream[i]);
	}

	hipEventDestroy(begin);
	hipEventDestroy(end);

	return (finish.QuadPart - start.QuadPart) * 1000 / (double)freq.QuadPart;
}

int menu()
{
	unsigned int width, height, channels;
	int image_choise = 1;
	char* input_file = nullptr;
	char* cpu_result_file = nullptr;
	char* gpu_result_file = nullptr;

	while (image_choise) {

		choise_image_and_format(&input_file, &cpu_result_file, &gpu_result_file);

		uint8_t* input_image = nullptr;

		if (!load_ppm(input_file, &input_image, &width, &height, &channels))
		{
			cout << "Error in loading file" << endl;
			return 1;
		}

		if (channels != 1 && channels != 3)
		{
			cout << "Error in count channels" << endl;
			free(input_image);
			return 1;
		}

		cout << endl << "Width " << width << endl << "Height " << height << endl << "Channels " << channels << endl;
		
		if (image_processing_menu(input_file, cpu_result_file, gpu_result_file, input_image, width, height, channels))
			return 1;

		free(input_image);

		cout << "Convert new image? 1 - yes, 0 - no" << endl;
		cin >> image_choise;
	}
	return 0;
}

int image_processing_menu(const char* input_file, const char* cpu_result_file, const char* gpu_result_file, 
						  uint8_t* input_image, const int width, const int height, const int channels)
{
	int choise = 1;

	while (choise)
	{
		change_filter();

		uint8_t* cpu_image = memory_alloc(width*height, channels);
		if (!cpu_image)
		{
			cout << "Error in memory allocation" << endl;
			free(input_image);
			return 1;
		}

		uint8_t* gpu_image = memory_alloc(width*height, channels);
		if (!gpu_convert_image)
		{
			cout << "Error in memory allocation" << endl;
			free(input_image);
			free(cpu_image);
			return 1;
		}

		cout << endl << "CPU processing" << endl;
		cout << "CPU time " << cpu_convert_image(input_image, cpu_image, width, height, channels) << endl;

		cout << endl << "Use cuda stream? 1-yes" << endl;
		cin >> choise;

		if (choise) {
			choise = 0;
			if (channels == 3)
			{
				cout << "Use cuda stream and cpu? 1-yes" << endl;
				cin >> choise;
			}
			cout << endl << "GPU processing" << endl;
			cout << "Time " << (choise == 1 ?
				cpu_and_gpu_convert_image(input_image, gpu_image, width, height, channels) :
				gpu_stream_convert_image(input_image, gpu_image, width, height, channels)) << endl;
		}
		else {
			cout << endl << "GPU processing" << endl;
			cout << "GPU time " << gpu_convert_image(input_image, gpu_image, width, height, channels) << endl;
		}

		int result = equals(cpu_image, gpu_image, width, height, channels);

		cout << ((result == -1) ? "Image is equals" : "Error in byte ");
		if (result != -1)
			cout << result << endl;
		else
			cout << endl;

		cout << endl << "Saving of results" << endl;

		if (!save_ppm(cpu_result_file, cpu_image, width, height, channels))
		{
			cout << "Error in save file" << endl;
			free(input_image);
			free(cpu_image);
			free(gpu_image);
			return 1;
		}

		if (!save_ppm(gpu_result_file, gpu_image, width, height, channels))
		{
			cout << "Error in save file" << endl;
			free(input_image);
			free(cpu_image);
			free(gpu_image);
			return 1;
		}
		free(cpu_image);
		free(gpu_image);

		cout << "Convert image again? 1 - yes, 0 - no" << endl;
		cin >> choise;
	}
	return 0;
}

void change_filter()
{
	show_filter();
	int choise = 0;
	cout << "Change filter? 1-yes, 0-no" << endl;
	cin >> choise;
	if (!choise)
		return;
	cout << "Select: 1 - enter yourself, 0 - select from specified" << endl;
	cin >> choise;
	if (!choise)
		choise_filter();
	else
		for (int i = 0; i < 3; i++)
			for (int j = 0; j < 3; j++)
			{
				cout << "Enter the number" << endl;
				cin >> filter[i][j];
			}

	show_filter();
}

void show_filter()
{
	cout << endl << "Filter:" << endl << endl;
	for (int i = 0; i < 3; i++)
	{
		for (int j = 0; j < 3; j++)
		{
			cout << filter[i][j] << " ";
		}
		cout << endl;
	}
	cout << endl;
}

void fill_filter(float new_filter[3][3])
{
	for (int i = 0; i < 3; i++)
	{
		for (int j = 0; j < 3; j++)
			filter[i][j] = new_filter[i][j];
	}
}

bool error(hipError_t val)
{
	if (!val)
		return false;
	cout << "Cuda Error " << hipGetLastError() << endl;
	return true;
}